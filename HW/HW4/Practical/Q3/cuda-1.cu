#include "hip/hip_runtime.h"

#include <iostream>
#include <math.h>
#include <string>

#include "opencv2/shape.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/imgproc.hpp"
#include <opencv2/core/utility.hpp>

using namespace std;
using namespace cv;

__global__
void invert(uchar *image, uchar *inverted_image, size_t total_size) {
    for (int i = 0; i < total_size; i++)
        inverted_image[i] = 255 - image[i];
}


int main(void) {
    hipEvent_t start, stop;
    float ms = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    string in_path = "pics/pic2.jpg";
    string out_path = "out.jpg";

    cout << "Reading image..." << endl;
    Mat M = imread(in_path);
    Size imgsize = M.size();

    Mat inverted_M(M.size(), M.type(), Scalar(0, 0, 0));

    size_t total_size = imgsize.width * imgsize.height * M.channels();
    int total_bytes = total_size * sizeof(uchar);

    uchar *image_host, *inverted_image_host;

    image_host = M.data;
    inverted_image_host = inverted_M.data;

    uchar *image, *inverted_image;

    hipMalloc(&image, total_bytes);
    hipMalloc(&inverted_image, total_bytes);

    hipMemcpy(image, image_host, total_bytes, hipMemcpyHostToDevice);
    hipMemcpy(inverted_image, inverted_image_host, total_bytes, hipMemcpyHostToDevice);

    cout << "inverting..." << endl;

    int blockSize = 1;
    int numBlocks = 1;

    hipEventRecord(start);
    invert<<<numBlocks, blockSize>>>(image, inverted_image, total_size);
    hipEventRecord(stop);

    hipDeviceSynchronize();

    hipEventElapsedTime(&ms, start, stop);
    cout << "Time: " << ms << "ms" << endl;

    hipHostMalloc(&inverted_image_host, total_bytes);

    hipMemcpy(inverted_image_host, inverted_image, total_bytes, hipMemcpyDeviceToHost);

    inverted_M = Mat(imgsize, M.type(), inverted_image_host);

    cout << "Writing image..." << endl;
    imwrite(out_path, inverted_M);

    hipFree(image);
    hipFree(inverted_image);

    cout << "DONE" << endl;
    return 0;
}
